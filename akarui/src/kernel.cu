#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_device_runtime_api.h"

#include <stdio.h>
#include <math.h>

#include <glm/glm.hpp>

#include "kernel.h"

__device__ bool intersectRayTriangle(glm::vec3 o, glm::vec3 d,
  glm::vec3 v0, glm::vec3 v1, glm::vec3 v2,
  glm::vec2& uv, float& t)
{
  glm::vec3 v0v1 = v1 - v0;
  glm::vec3 v0v2 = v2 - v0;
  glm::vec3 pvec = glm::cross(d, v0v2);
  float det = glm::dot(v0v1, pvec);
#ifdef CULLING 
  // if the determinant is negative the triangle is backfacing
  // if the determinant is close to 0, the ray misses the triangle
  if (det < FLT_EPSILON) return false;
#else 
  // ray and triangle are parallel if det is close to 0
  if (fabs(det) < FLT_EPSILON) return false;
#endif 
  float invDet = 1 / det;

  glm::vec3 tvec = o - v0;
  uv.x = glm::dot(tvec, pvec) * invDet;
  if (uv.x < 0.0f || uv.x > 1.0f) return false;

  glm::vec3 qvec = glm::cross(tvec, v0v1);
  uv.y = glm::dot(d, qvec) * invDet;
  if (uv.y < 0.0f || uv.x + uv.y > 1.0f) return false;

  t = glm::dot(v0v2, qvec) * invDet;

  return true;
}

__device__ bool rayIntersectSphere(glm::vec3 origin, glm::vec3 direction, glm::vec4 sphere, glm::vec2& intersectionPoints) {
  float c = glm::length(origin - glm::vec3(sphere)) - sphere.w*sphere.w;
  float dotVal = glm::dot(direction, (origin - glm::vec3(sphere)));
  float sqrtVal = dotVal*dotVal - glm::dot(origin - glm::vec3(sphere), origin - glm::vec3(sphere)) + sphere.w*sphere.w;
  if (sqrtVal <= 0.0) {
    return false;
  }
  if (sqrtVal == 0.0) {
    intersectionPoints.x = -dotVal;
    intersectionPoints.y = -dotVal;
    return true;
  }
  else {
    float d1 = -(dotVal)+sqrt(sqrtVal);
    float d2 = -(dotVal)-sqrt(sqrtVal);
    if (d1 < 0.0 && d2 < 0.0) {
      return false;
    }
    else if (d1 < 0.0 || d2 < 0.0) {
      intersectionPoints.x = glm::min(d1, d2);
      intersectionPoints.y = glm::max(d1, d2);
      return true;
    }
    else {
      intersectionPoints.x = glm::min(d1, d2);
      intersectionPoints.y = glm::max(d1, d2);
      return true;
    }
  }
}

__device__ void traceScene(glm::vec3 origin, glm::vec3 dir, const Scene* scene, float& t, glm::vec2& hitUV, int& hitFace, const Mesh*& hitMesh, const Material*& hitMat)
{
  for (int meshIdx = 0; meshIdx < scene->meshCount; ++meshIdx) {
    Mesh* mesh = scene->meshes[meshIdx];
    for (int tri = 0; tri < mesh->idxCount/3; ++tri) {
      int a = mesh->idx[tri*3];
      int b = mesh->idx[tri*3+1];
      int c = mesh->idx[tri*3+2];

      glm::vec3 v0 = mesh->pos[a];
      glm::vec3 v1 = mesh->pos[b];
      glm::vec3 v2 = mesh->pos[c];

      glm::vec2 uv;
      float hitT = INFINITY;
      intersectRayTriangle(origin, dir, v0, v1, v2, uv, hitT);
      if (hitT != INFINITY && hitT < t && hitT >= 0.0f) {
        t = hitT;
        hitFace = tri;
        hitUV = uv;
        hitMesh = mesh;
        hitMat = &scene->defaultMat;
      }
    }
  }
}

__global__ void renderPixel(hipSurfaceObject_t surface, dim3 screenRes, dim3 blockSize, float time, Scene* scene, glm::vec3 camPos, glm::mat4 viewRot)
{
    int x = blockIdx.x * blockSize.x + threadIdx.x;
    int y = blockIdx.y * blockSize.y + threadIdx.y;

    float aspect = (float)screenRes.y / screenRes.x;

    glm::vec2 normalisedCoord = 2.0f * glm::vec2(x, screenRes.y - y) / glm::vec2(screenRes.x, screenRes.y) - glm::vec2(1.0f);
    normalisedCoord.y *= aspect;

    glm::vec3 origin = camPos;
    glm::vec3 direction = glm::mat3(viewRot) * glm::normalize(glm::vec3(normalisedCoord.x, normalisedCoord.y, -1.0f));

    // test intersection with each tri
    float minT = INFINITY;
    glm::vec2 hitUV = glm::vec2(0.0f);
    int hitFace = -1;
    const Mesh* hitMesh = nullptr;
    const Material* hitMat = &scene->defaultMat;

    // raytrace the lights for debug drawing
    for (int i = 0; i < scene->lightCount; ++i) {
      Light* light = &scene->lights[i];
      if (light->type == Light::Type_Point) {
        glm::vec2 hit;
        if (rayIntersectSphere(origin, direction, glm::vec4(light->pos + glm::vec3(sin(time), cos(time), 0.0f) * 0.5f, 0.1f), hit))
          minT = glm::min(hit.x, minT);
      }
    }

    // raytrace the scene
    traceScene(origin, direction, scene, minT, hitUV, hitFace, hitMesh, hitMat);

    glm::vec4 outColour;

    if (minT != INFINITY) {
      glm::vec3 normal = glm::vec3(1.0f);

      if (hitMesh != nullptr && hitFace >= 0) {
        // we got a hit. calculate the normal
        int a = hitMesh->idx[hitFace * 3 + 0];
        int b = hitMesh->idx[hitFace * 3 + 1];
        int c = hitMesh->idx[hitFace * 3 + 2];
        glm::vec3 v0(hitMesh->pos[a]), v1(hitMesh->pos[b]), v2(hitMesh->pos[c]);
        normal = glm::normalize(glm::cross(v1 - v0, v1 - v2));
      }

      glm::vec3 hitPoint = origin + minT * direction;

      glm::vec3 light = scene->Ia;

      for (int i = 0; i < scene->lightCount; ++i) {
        Light* l = &scene->lights[i];

        glm::vec3 Lm;

        if (l->type == l->Type_Point) {
          glm::vec3 lightDiff = hitPoint - (l->pos + glm::vec3(sin(time), cos(time), 0.0f) * 0.5f);
          float lightDist = glm::length(lightDiff);
          Lm = lightDiff / lightDist;
        }
        else if (l->type == l->Type_Directional) {
          Lm = l->dir;
        }

        // Shadow ray
        float hit = INFINITY;
        glm::vec2 tmp0; int tmp1; Mesh* tmp2;
        //traceScene(hitPoint + normal * 0.1f, Lm, scene, hit, tmp0, tmp1, tmp2);

        if (hit == INFINITY) {
          // lambert
          light += glm::dot(normal, Lm);

          // blinn-phong
        }
      }
      
      outColour = glm::clamp(glm::vec4(light, 1.0f), 0.0f, 1.0f);
    }
    else {
      outColour = glm::vec4(normalisedCoord, 0.0f, 1.0f);
      //outColour = glm::vec4(0.0f, 0.0f, 0.0f, 1.0f);
    }

    if (x < screenRes.x && y < screenRes.y)
      surf2Dwrite(float4{outColour.x, outColour.y, outColour.z, outColour.w}, surface, x * 4 * sizeof(float), y);
}

hipError_t renderScreen(hipSurfaceObject_t surface, dim3 screenRes, float time, Scene* scene, glm::vec3 camPos, glm::mat4& viewMat)
{
    hipError_t cudaStatus = hipSuccess;

    // calculate occupancy
    int recBlockSize, minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &recBlockSize, renderPixel, 0, 0);

    // Convert block size to 2d
    dim3 blockSize(1, recBlockSize);
    while (blockSize.x < blockSize.y) {
      blockSize.x *= 2;
      blockSize.y /= 2;
    }

    dim3 gridSize(int(ceil(screenRes.x/float(blockSize.x))), int(ceil(screenRes.y/float(blockSize.y))));

    renderPixel<<<gridSize, blockSize>>>(surface, screenRes, blockSize, time, scene, camPos, viewMat);

    //// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    
    return cudaStatus;
}
