
#include "hip/hip_runtime.h"

#include "cuda_device_runtime_api.h"

#include <stdio.h>
#include <math.h>

hipError_t renderScreen(hipSurfaceObject_t, dim3 screen_res);

__global__ void renderPixel(hipSurfaceObject_t surface, dim3 screenRes, dim3 blockSize)
{
    int x = blockIdx.x * blockSize.x + threadIdx.x;
    int y = blockIdx.y * blockSize.y + threadIdx.y;
    //if (y != 0)
    //printf("running thread %d %d\n", x, y);
    float4 pixel = {x/float(screenRes.x*screenRes.y), x/float(screenRes.x), y/float(screenRes.y), 0.0f};
    if (x < screenRes.x && y < screenRes.y)
      surf2Dwrite(pixel, surface, x * 4 * sizeof(float), y);
}

int runkernel(hipSurfaceObject_t cudaSurfaceObject, dim3 screen_res)
{
  // Add vectors in parallel.
  hipError_t cudaStatus = renderScreen(cudaSurfaceObject, screen_res);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "renderScreen failed!");
      return 1;
  }

  return 0;
}

hipError_t renderScreen(hipSurfaceObject_t surface, dim3 screenRes)
{
    hipError_t cudaStatus = hipSuccess;

    // calculate occupancy
    int recBlockSize, minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &recBlockSize, renderPixel, 0, 0);

    // Convert block size to 2d
    dim3 blockSize(1, recBlockSize);
    while (blockSize.x < blockSize.y) {
      blockSize.x *= 2;
      blockSize.y /= 2;
    }

    dim3 gridSize(int(ceil(screenRes.x/float(blockSize.x))), int(ceil(screenRes.y/float(blockSize.y))));

    renderPixel<<<gridSize, blockSize>>>(surface, screenRes, blockSize);

    //// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    //// cudaDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    
    return cudaStatus;
}
