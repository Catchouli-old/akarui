#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_device_runtime_api.h"

#include <stdio.h>
#include <math.h>

#include <glm/glm.hpp>

#include "kernel.h"

__device__ bool intersectRayTriangle(glm::vec3 o, glm::vec3 d,
  glm::vec3 v0, glm::vec3 v1, glm::vec3 v2,
  glm::vec2& uv, float& t)
{
  glm::vec3 v0v1 = v1 - v0;
  glm::vec3 v0v2 = v2 - v0;
  glm::vec3 pvec = glm::cross(d, v0v2);
  float det = glm::dot(v0v1, pvec);
#ifdef CULLING 
  // if the determinant is negative the triangle is backfacing
  // if the determinant is close to 0, the ray misses the triangle
  if (det < FLT_EPSILON) return false;
#else 
  // ray and triangle are parallel if det is close to 0
  if (fabs(det) < FLT_EPSILON) return false;
#endif 
  float invDet = 1 / det;

  glm::vec3 tvec = o - v0;
  uv.x = glm::dot(tvec, pvec) * invDet;
  if (uv.x < 0.0f || uv.x > 1.0f) return false;

  glm::vec3 qvec = glm::cross(tvec, v0v1);
  uv.y = glm::dot(d, qvec) * invDet;
  if (uv.y < 0.0f || uv.x + uv.y > 1.0f) return false;

  t = glm::dot(v0v2, qvec) * invDet;

  return true;
}

__device__ void traceScene(glm::vec3 origin, glm::vec3 dir, const Scene* scene, float& t, glm::vec2& hitUV, int& hitFace, Mesh*& hitMesh)
{
  for (int meshIdx = 0; meshIdx < scene->meshCount; ++meshIdx) {
    Mesh* mesh = scene->meshes[meshIdx];
    for (int tri = 0; tri < mesh->idxCount/3; ++tri) {
      int a = mesh->idx[tri*3];
      int b = mesh->idx[tri*3+1];
      int c = mesh->idx[tri*3+2];

      glm::vec3 v0 = mesh->pos[a];
      glm::vec3 v1 = mesh->pos[b];
      glm::vec3 v2 = mesh->pos[c];

      glm::vec2 uv;
      float hitT = INFINITY;
      intersectRayTriangle(origin, dir, v0, v1, v2, uv, hitT);
      if (hitT != INFINITY && hitT < t && hitT >= 0.0f) {
        t = hitT;
        hitFace = tri;
        hitUV = uv;
        hitMesh = mesh;
      }
    }
  }
}

__global__ void renderPixel(hipSurfaceObject_t surface, dim3 screenRes, dim3 blockSize, float time, Scene* scene, glm::vec3 camPos, glm::mat4 viewRot)
{
    int x = blockIdx.x * blockSize.x + threadIdx.x;
    int y = blockIdx.y * blockSize.y + threadIdx.y;

    float aspect = (float)screenRes.y / screenRes.x;

    glm::vec2 normalisedCoord = 2.0f * glm::vec2(x, screenRes.y - y) / glm::vec2(screenRes.x, screenRes.y) - glm::vec2(1.0f);
    normalisedCoord.y *= aspect;

    glm::vec3 origin = camPos;
    glm::vec3 direction = glm::mat3(viewRot) * glm::normalize(glm::vec3(normalisedCoord.x, normalisedCoord.y, 1.0f));

    // test intersection with each tri
    float minT = INFINITY;
    glm::vec2 hitUV;
    int hitFace;
    Mesh* hitMesh;

    traceScene(origin, direction, scene, minT, hitUV, hitFace, hitMesh);

    glm::vec4 outColour;

    if (minT != INFINITY) {
      // we got a hit. calculate the normal
      int a = hitMesh->idx[hitFace*3+0];
      int b = hitMesh->idx[hitFace*3+1];
      int c = hitMesh->idx[hitFace*3+2];
      glm::vec3 v0(hitMesh->pos[a]), v1(hitMesh->pos[b]), v2(hitMesh->pos[c]);
      glm::vec3 normal = glm::normalize(glm::cross(v1 - v0, v1 - v2));

      glm::vec3 hitPoint = origin + minT * direction;

      // calculate lighting using a simple ambient + lambert + blinn-phong BRDF
      const Material defaultMaterial { Material::Type_Constant
                                     , glm::vec3(1.0f, 0.0f, 1.0f)
                                     , glm::vec3(1.0f, 0.0f, 1.0f)
                                     , glm::vec3(1.0f, 0.0f, 1.0f)
                                     , 1.0f
                                     };

      const Material* mat = &defaultMaterial;

      glm::vec3 light = scene->Ia;

      for (int i = 0; i < scene->lightCount; ++i) {
        Light* l = &scene->lights[i];

        glm::vec3 lightDiff = l->pos - hitPoint;
        float lightDist = glm::length(lightDiff);
        glm::vec3 Lm = lightDiff / lightDist;

        // Shadow ray
        float hit = INFINITY;
        glm::vec2 tmp0; int tmp1; Mesh* tmp2;
        //traceScene(hitPoint + normal * 0.1f, Lm, scene, hit, tmp0, tmp1, tmp2);

        if (hit == INFINITY) {
          // lambert
          light += glm::dot(normal, Lm) * l->Id * mat->Kd;

          // blinn-phong
        }
      }
      
      outColour = glm::vec4(light, 1.0f);
    }
    else {
      outColour = glm::vec4(normalisedCoord, 0.0f, 1.0f);
      //outColour = glm::vec4(0.0f, 0.0f, 0.0f, 1.0f);
    }

    if (x < screenRes.x && y < screenRes.y)
      surf2Dwrite(float4{outColour.x, outColour.y, outColour.z, outColour.w}, surface, x * 4 * sizeof(float), y);
}

hipError_t renderScreen(hipSurfaceObject_t surface, dim3 screenRes, float time, Scene* scene, glm::vec3 camPos, glm::mat4& viewMat)
{
    hipError_t cudaStatus = hipSuccess;

    // calculate occupancy
    int recBlockSize, minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &recBlockSize, renderPixel, 0, 0);

    // Convert block size to 2d
    dim3 blockSize(1, recBlockSize);
    while (blockSize.x < blockSize.y) {
      blockSize.x *= 2;
      blockSize.y /= 2;
    }

    dim3 gridSize(int(ceil(screenRes.x/float(blockSize.x))), int(ceil(screenRes.y/float(blockSize.y))));

    renderPixel<<<gridSize, blockSize>>>(surface, screenRes, blockSize, time, scene, camPos, viewMat);

    //// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    
    return cudaStatus;
}
