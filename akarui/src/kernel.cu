
#include "hip/hip_runtime.h"

#include "cuda_device_runtime_api.h"

#include <stdio.h>

hipError_t renderScreen(hipSurfaceObject_t);

__global__ void renderPixel(hipSurfaceObject_t surface)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    //if (y != 0)
    //printf("running thread %d %d\n", x, y);
    surf2Dwrite(float4{x/(800.0f*600.0f), x/800.0f, y/600.0f, 0.0f}, surface, x * 4 * sizeof(float), y);
}

int runkernel(hipSurfaceObject_t cudaSurfaceObject)
{
  // Add vectors in parallel.
  hipError_t cudaStatus = renderScreen(cudaSurfaceObject);
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "renderScreen failed!");
      return 1;
  }

  return 0;
}

hipError_t renderScreen(hipSurfaceObject_t surface)
{
    hipError_t cudaStatus = hipSuccess;

    // calculate occupancy
    int blockSize, minGridSize;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, renderPixel, 0, 0);

    // Launch a kernel on the GPU with one thread for each element.
    dim3 dimBlock(1, 1);
    dim3 dimGrid(800, 600);
    renderPixel<<<dimGrid, dimBlock>>>(surface);

    //// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    //// cudaDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    
    return cudaStatus;
}
